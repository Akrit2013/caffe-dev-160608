#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/scaleinvariantlog_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {



template <typename Dtype>
__global__ void Forward_gpu_kernel(
		 const int nthreads,
		 const Dtype* const data_label,
		 Dtype* data_diff,
		 Dtype* data_pred,
		 Dtype* bad_pixel_data,
		 const int num,
		 const int channels,
		 const int height,
		 const int width,
		 const Dtype max_label,
		 const Dtype min_label){
	CUDA_KERNEL_LOOP(index, nthreads){
		const int n = index / height;
		const int h = index % height;

		const int data_offset = (n*channels*height+h)*width;
		const int bad_pixel_idx = index;
		const int interval = height * width;

		// Iter the width and channels
		for (int w = 0; w < width; w++){
			// Iter the channels
			int err_counter = 0;
			for (int c = 0; c < channels; c++){
				const int idx = data_offset + c * interval + w;
				// Check the boundary of the prediction
				if(data_pred[idx] > max_label){
					data_pred[idx] = max_label;
					data_diff[idx] = log(data_pred[idx]) - log(data_label[idx]);
				}else if(data_pred[idx] < min_label){
					data_pred[idx] = min_label;
					data_diff[idx] = log(data_pred[idx]) - log(data_label[idx]);
				}

				if (data_label[idx] > max_label){
					err_counter++;
				}else if(data_label[idx] < min_label){
					err_counter++;
				}
			}

			// Only if all channels invalid, the pixel will be considered
			// as invalid
			if(err_counter == channels){
				bad_pixel_data[bad_pixel_idx] += channels;
				for (int c = 0; c < channels; c++){
					const int idx = data_offset + c * interval + w;
					data_diff[idx] = 0;
				}
			}
		}
	}
}


template <typename Dtype>
__global__ void Backward_gpu_kernel(
		const int nthreads,
		const Dtype* const diff_data,
		const Dtype* const pred_data,
		Dtype* bottom_diff,
		const Dtype* const vecValidPixelNum_data,
		const Dtype* const vecSum_data,
		const int num,
		const int channels,
		const int height,
		const int width,
		const int sign,
		const int valid_pixel_num,
		const Dtype delta){

	CUDA_KERNEL_LOOP(index, nthreads){
		const int n = index / height / channels;
		const int c = (index / height) % channels;
		const int h = index % height;

		const int data_offset = ((n*channels + c)*height + h)*width;

		// bottom diff = sign * (w1 * diff - w2) / pred
		// Calc the w1
		const Dtype w1 = Dtype(1) / valid_pixel_num;
		// Calc the w2
		const Dtype valid_sum = vecSum_data[n];
		const Dtype valid_num = vecValidPixelNum_data[n];
		const Dtype w2 = valid_sum * delta / num / valid_num / valid_num;

		for(int w = 0; w < width; w++){
			bottom_diff[data_offset+w] = sign * (w1 * diff_data[data_offset+w] - w2) / pred_data[data_offset+w];
		}
	}
}

template <typename Dtype>
void ScaleInvariantLogLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_log(count, bottom[0]->gpu_data(), pred_log_.mutable_gpu_data());
  caffe_gpu_log(count, bottom[1]->gpu_data(), label_log_.mutable_gpu_data());

  caffe_gpu_sub(
      count,
	  pred_log_.gpu_data(),
	  label_log_.gpu_data(),
      diff_.mutable_gpu_data());

  Dtype* data_diff = diff_.mutable_gpu_data();
  Dtype* vecValidPixelNum_data = vecValidPixelNum_.mutable_cpu_data();
  Dtype* vecSum_data = vecSum_.mutable_cpu_data();
  Dtype* data_pred = bottom[0]->mutable_gpu_data(); 
  const Dtype* data_label = bottom[1]->gpu_data();
  const int num = bottom[0]->num();
  const int channels = bottom[0]->channels();
  const int height = bottom[0]->height();
  const int width = bottom[0]->width();
  // Set the number of the kernel]
  const int num_kernels = num * height;
  // Set the bad_pixel_ buffer to 0
  Dtype* bad_pixel_data = bad_pixel_.mutable_gpu_data();
  caffe_gpu_set(bad_pixel_.count(), Dtype(0), bad_pixel_data);
  
  // Find the bad pixel and alter the diff
  // Also check the prediction, if the prediction out of the scope, set the value
  // into the scope
  if(is_use_bad_pixel_ == true){
	  Forward_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
			  num_kernels,
			  data_label,
			  data_diff,
			  data_pred,
			  bad_pixel_data,
			  num,
			  channels,
			  height,
			  width,
			  max_val_,
			  min_val_);
  }
  // The pixel number per image
  Dtype pixel_num = bottom[0]->count(1);
  Dtype bad_pixel_count;
  // Calc the whole valid pixel number
  if (is_adjust_pixel_num_){
	  caffe_gpu_asum(bad_pixel_.count(), bad_pixel_data, &bad_pixel_count);
	  valid_pixel_num_ = count - bad_pixel_count;
  }else{
	  valid_pixel_num_ = count;
  }
  // Calc the each image's valid pixel number in minibatch
  for (int n = 0; n < diff_.num(); n++){
	  if (is_adjust_pixel_num_){
		  Dtype val;
		  int offset = bad_pixel_.offset(n);
		  caffe_gpu_asum(height, bad_pixel_data + offset, &val);
		  vecValidPixelNum_data[n] = pixel_num - val;
	  }else{
		  vecValidPixelNum_data[n] = pixel_num;
	  }
  }

  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / valid_pixel_num_ / Dtype(2);

  // Calc the second term of the loss
  for (int n = 0; n < bottom[0]->num(); n++){
	  const Dtype* cdata_diff = diff_.cpu_data() + diff_.offset(n);
	  Dtype valid_num = vecValidPixelNum_data[n];
	  Dtype vecSum = caffe_cpu_sum(pixel_num, cdata_diff);
	  vecSum_data[n] = vecSum;
	  loss += vecSum_data[n] * vecSum_data[n] / valid_num / valid_num / bottom[0]->num() * delta_ / Dtype(2);
  }

  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void ScaleInvariantLogLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
//      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
//      caffe_gpu_axpby(
//          bottom[i]->count(),              // count
//          alpha,                              // alpha
//          diff_.gpu_data(),                   // a
//          Dtype(0),                           // beta
//          bottom[i]->mutable_gpu_diff());  // b
//    }
	const Dtype* diff_data = diff_.gpu_data();
	const Dtype* vecValidPixelNum_data = vecValidPixelNum_.gpu_data();
	const Dtype* vecSum_data = vecSum_.gpu_data();
	const Dtype* pred_data = bottom[i]->gpu_data();
	Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
	const int num = bottom[i]->num();
	const int channels = bottom[i]->channels();
	const int height = bottom[i]->height();
	const int width = bottom[i]->width();

	const int num_kernels = num * channels * height;
	Backward_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
			num_kernels,
			diff_data,
			pred_data,
			bottom_diff,
			vecValidPixelNum_data,
			vecSum_data,
			num,
			channels,
			height,
			width,
			sign,
			valid_pixel_num_,
			delta_);

  }
}
}

INSTANTIATE_LAYER_GPU_FUNCS(ScaleInvariantLogLossLayer);

}  // namespace caffe
